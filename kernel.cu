
#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 16

__global__ void mysgemm(int m, int n, int p, const float *A, const float *B, float *C) {
    __shared__ float h[TILE_SIZE][TILE_SIZE], v[TILE_SIZE][TILE_SIZE];

    int carry = int(p / TILE_SIZE);
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float ts = 0;

    for (int pos = 0; pos <= carry; pos++) {
        if ((row < m) && (pos * TILE_SIZE + threadIdx.x < p)) {
            h[threadIdx.y][threadIdx.x] = A[row * p + pos * TILE_SIZE + threadIdx.x];
        } else {
            h[threadIdx.y][threadIdx.x] = 0;
        }

        if ((col < n) && (pos * TILE_SIZE + threadIdx.y < p)) {
            v[threadIdx.y][threadIdx.x] = B[(pos * TILE_SIZE + threadIdx.y) * n + col];
        } else {
            v[threadIdx.y][threadIdx.x] = 0;
        }

        __syncthreads();

        if (row < m && col < n) {
            for (int idx = 0; idx < TILE_SIZE; idx++) {
                ts += h[threadIdx.y][idx] * v[idx][threadIdx.x];
            }
        }

        __syncthreads();
    }

    if (row < m && col < n) {
        C[row * n + col] += ts;
    }
}

void basicSgemm(int m, int n, int k, const float *A, const float *B, float *C) {
    const unsigned int BLOCK_SIZE = TILE_SIZE;
    dim3 DimGrid((n - 1) / BLOCK_SIZE + 1, (m - 1) / BLOCK_SIZE + 1, 1);
    dim3 DimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
    mysgemm<<<DimGrid, DimBlock>>>(m, n, k, A, B, C);
}
